#include <torch/extension.h>
#include <ATen/native/cuda/Loops.cuh>
#include "../cbesk.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

namespace torch_bessel {

at::Tensor besselk_cpu(const at::Tensor& nu, const at::Tensor& z) {
  TORCH_CHECK(nu.scalar_type() == at::kFloat);
  TORCH_CHECK(z.scalar_type() == at::kComplexFloat);
  TORCH_INTERNAL_ASSERT(nu.device().type() == at::DeviceType::CUDA);
  TORCH_INTERNAL_ASSERT(z.device().type() == at::DeviceType::CUDA);
  at::Tensor result = torch::empty(at::IntArrayRef(), at::device(at::kCUDA).dtype(at::kComplexFloat)).resize_(0);
  auto iter = (
    at::TensorIteratorConfig()
    .set_check_mem_overlap(true)
    .allow_cpu_scalars(true)
    .add_output(result)
    .add_input(nu)
    .add_input(z)
  ).build();
  at::native::gpu_kernel(iter, []GPU_LAMBDA(float nu, c10::complex<float> z) -> c10::complex<float> {
      return cbesk(nu, z);
  });
  return result;
}

// Registers CUDA implementations for besselk
TORCH_LIBRARY_IMPL(torch_bessel, CUDA, m) {
  m.impl("besselk", &besselk_cuda);
}

}
