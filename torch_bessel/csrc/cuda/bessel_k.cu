#include <torch/extension.h>
#include <ATen/TensorIterator.h>
#include <ATen/native/cuda/Loops.cuh>
#include <c10/util/complex.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/tuple.h>

#include "../bessel_k.h"
#include "../iterator.h"

namespace torch_bessel {

at::Tensor bessel_k0_forward_cuda(const at::Tensor& z) {
  TORCH_INTERNAL_ASSERT(z.device().type() == at::DeviceType::CUDA);
  at::ScalarType dtype = z.scalar_type();
  at::Tensor result = torch::empty(at::IntArrayRef(), at::device(at::kCUDA).dtype(dtype)).resize_(0);
  at::TensorIterator iter = build_iterator_11(result, z);
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES(dtype, "bessel_k0_forward_cuda", [&]() {
    at::native::gpu_kernel(iter, []GPU_LAMBDA(scalar_t z) -> scalar_t {
      return bessel_k0_forward(z);
    });
  });
  return result;
}

std::tuple<at::Tensor, at::Tensor> bessel_k0_forward_backward_cuda(const at::Tensor& z) {
  TORCH_INTERNAL_ASSERT(z.device().type() == at::DeviceType::CUDA);
  at::ScalarType dtype = z.scalar_type();
  at::Tensor result1 = torch::empty(at::IntArrayRef(), at::device(at::kCUDA).dtype(dtype)).resize_(0);
  at::Tensor result2 = torch::empty(at::IntArrayRef(), at::device(at::kCUDA).dtype(dtype)).resize_(0);
  at::TensorIterator iter = build_iterator_21(result1, result2, z);
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES(dtype, "bessel_k0_forward_backward_cuda", [&]() {
    at::native::gpu_kernel_multiple_outputs(iter, []GPU_LAMBDA(scalar_t z) -> thrust::tuple<scalar_t, scalar_t> {
      scalar_t cy[2];
      bessel_k0_forward_backward(z, cy);
      return thrust::make_tuple(cy[0], cy[1]);
    });
  });
  return std::make_tuple(result1, result2);
}

// Registers CUDA implementations for bessel_k
TORCH_LIBRARY_IMPL(torch_bessel, CUDA, m) {
  m.impl("bessel_k0_forward", &bessel_k0_forward_cuda);
  m.impl("bessel_k0_forward_backward", &bessel_k0_forward_backward_cuda);
}

}
