#include <torch/extension.h>
#include <ATen/TensorIterator.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/OpMathType.h>
#include <c10/util/complex.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/tuple.h>

#include "../bessel_k.h"
#include "../iterator.h"

namespace torch_bessel {

at::Tensor modified_bessel_k0_complex_forward_cuda(const at::Tensor& z) {
  TORCH_INTERNAL_ASSERT(z.device().type() == at::DeviceType::CUDA);
  at::ScalarType dtype = z.scalar_type();
  at::Tensor result = torch::empty(at::IntArrayRef(), at::device(at::kCUDA).dtype(dtype)).resize_(0);
  at::TensorIterator iter = build_iterator_11(result, z);
  AT_DISPATCH_COMPLEX_TYPES_AND(at::kComplexHalf, dtype, "modified_bessel_k0_complex_forward_cuda", [&]() {
    at::native::gpu_kernel(iter, []GPU_LAMBDA(scalar_t z) -> scalar_t {
      using opmath_t = at::opmath_type<scalar_t>;
      return modified_bessel_k0_complex_forward(static_cast<opmath_t>(z));
    });
  });
  return result;
}

std::tuple<at::Tensor, at::Tensor> modified_bessel_k0_complex_forward_backward_cuda(const at::Tensor& z) {
  TORCH_INTERNAL_ASSERT(z.device().type() == at::DeviceType::CUDA);
  at::ScalarType dtype = z.scalar_type();
  at::Tensor result1 = torch::empty(at::IntArrayRef(), at::device(at::kCUDA).dtype(dtype)).resize_(0);
  at::Tensor result2 = torch::empty(at::IntArrayRef(), at::device(at::kCUDA).dtype(dtype)).resize_(0);
  at::TensorIterator iter = build_iterator_21(result1, result2, z);
  AT_DISPATCH_COMPLEX_TYPES_AND(at::kComplexHalf, dtype, "modified_bessel_k0_complex_forward_backward_cuda", [&]() {
    at::native::gpu_kernel_multiple_outputs(iter, []GPU_LAMBDA(scalar_t z) -> thrust::tuple<scalar_t, scalar_t> {
      using opmath_t = at::opmath_type<scalar_t>;
      opmath_t cy[2];
      modified_bessel_k0_complex_forward_backward(static_cast<opmath_t>(z), cy);
      return thrust::make_tuple(cy[0], cy[1]);
    });
  });
  return std::make_tuple(result1, result2);
}

at::Tensor modified_bessel_k1_complex_forward_cuda(const at::Tensor& z) {
  TORCH_INTERNAL_ASSERT(z.device().type() == at::DeviceType::CUDA);
  at::ScalarType dtype = z.scalar_type();
  at::Tensor result = torch::empty(at::IntArrayRef(), at::device(at::kCUDA).dtype(dtype)).resize_(0);
  at::TensorIterator iter = build_iterator_11(result, z);
  AT_DISPATCH_COMPLEX_TYPES_AND(at::kComplexHalf, dtype, "modified_bessel_k1_complex_forward_cuda", [&]() {
    at::native::gpu_kernel(iter, []GPU_LAMBDA(scalar_t z) -> scalar_t {
      using opmath_t = at::opmath_type<scalar_t>;
      return modified_bessel_k1_complex_forward(static_cast<opmath_t>(z));
    });
  });
  return result;
}

// Registers CUDA implementations for bessel_k
TORCH_LIBRARY_IMPL(torch_bessel, CUDA, m) {
  m.impl("modified_bessel_k0_complex_forward", &modified_bessel_k0_complex_forward_cuda);
  m.impl("modified_bessel_k0_complex_forward_backward", &modified_bessel_k0_complex_forward_backward_cuda);
  m.impl("modified_bessel_k1_complex_forward", &modified_bessel_k1_complex_forward_cuda);
}

}
