#include <torch/extension.h>
#include <ATen/TensorIterator.h>
#include <ATen/native/cuda/Loops.cuh>
#include <c10/util/complex.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "../bessel.h"
#include "../iterator.h"

namespace torch_bessel {

at::Tensor bessel_k_forward_cuda(const at::Tensor& v, const at::Tensor& z) {
  TORCH_INTERNAL_ASSERT(v.device().type() == at::DeviceType::CUDA);
  TORCH_INTERNAL_ASSERT(z.device().type() == at::DeviceType::CUDA);
  at::ScalarType dtype = at::promote_types(v.scalar_type(), z.scalar_type());
  at::Tensor result = torch::empty(at::IntArrayRef(), at::device(at::kCUDA).dtype(dtype)).resize_(0);
  at::TensorIterator iter = build_iterator(result, v, z);
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES(iter.common_dtype(), "bessel_k_forward_cuda", [&]() {
    at::native::gpu_kernel(iter, []GPU_LAMBDA(scalar_t v, scalar_t z) -> scalar_t {
        return bessel_k_forward(std::real(v), z);
    });
  });
  return result;
}

std::tuple<at::Tensor, at::Tensor> bessel_k_forward_backward_cuda(const at::Tensor& v, const at::Tensor& z) {
  TORCH_INTERNAL_ASSERT(v.device().type() == at::DeviceType::CUDA);
  TORCH_INTERNAL_ASSERT(z.device().type() == at::DeviceType::CUDA);
  at::ScalarType dtype = at::promote_types(v.scalar_type(), z.scalar_type());
  at::Tensor result1 = torch::empty(at::IntArrayRef(), at::device(at::kCUDA).dtype(dtype)).resize_(0);
  at::Tensor result2 = torch::empty(at::IntArrayRef(), at::device(at::kCUDA).dtype(dtype)).resize_(0);
  at::TensorIterator iter = build_iterator_2(result1, result2, v, z);
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES(iter.common_dtype(), "bessel_k_forward_backward_cuda", [&]() {
    at::native::gpu_kernel_multiple_outputs(iter, []GPU_LAMBDA(scalar_t v, scalar_t z) -> std::tuple<scalar_t, scalar_t> {
        return bessel_k_forward_backward(std::real(v), z);
    });
  });
  return std::make_tuple(result1, result2);
}

// Registers CUDA implementations for bessel_k
TORCH_LIBRARY_IMPL(torch_bessel, CUDA, m) {
  m.impl("bessel_k_forward", &bessel_k_forward_cuda);
  m.impl("bessel_k_forward_backward", &bessel_k_forward_backward_cuda);
}

}
